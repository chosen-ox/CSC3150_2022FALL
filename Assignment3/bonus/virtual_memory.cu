﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    // vm->invert_page_table[i + vm->PAGE_ENTRIES] = -1;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES, u32* LRU_ARRAY) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;
  vm->LRU_ARRAY = LRU_ARRAY;
  vm->time_counter = 0;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->LRU_ARRAY[i] = vm->time_counter++;
  }

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
/* Complete vm_write function to write value into data buffer */
  u32 offset = addr & 0x1f;
  u32 vpn = (addr & 0x0fffffff) >> 5;
  // u32 vpn = addr / 32;
  // printf("vpn%d\n", vpn);
  int page_entry = vm_search_vpn(vm, vpn);
  // printf("search %d", vm_search_vpn(vm, 3072));
  if (page_entry == -1) { 
    ++(*vm->pagefault_num_ptr);
    int lru_idx = vm_get_LRU_idx(vm);

    if ((vm->invert_page_table[lru_idx] & 0x80000000) == 0) {
      // printf("index%d\n",lru_idx);
      vm_swap_to_storage(vm, vm->invert_page_table[lru_idx], lru_idx);
    }
    vm_swap_to_data(vm, vpn, lru_idx);
    page_entry = vm_search_vpn(vm, vpn);
  }
  vm_update_queue(vm, page_entry);
  
  // printf("vpn:%d vm0%x  phy:%d, offset:%d\n", vpn, vm->invert_page_table[0], page_entry,offset);

  return vm->buffer[page_entry * vm->PAGESIZE + offset]; //TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  u32 offset = addr & 0x1f;
  u32 vpn = (addr & 0x0fffffff) >> 5;
  // u32 vpn =  addr >> 5;
  // printf("vpn%d\n", vpn);
  int page_entry = vm_search_vpn(vm, vpn);
  if (page_entry == -1) { 
    ++(*vm->pagefault_num_ptr);
    int lru_idx = vm_get_LRU_idx(vm);


    if ((vm->invert_page_table[lru_idx] & 0x80000000) == 0) {
      vm_swap_to_storage(vm, vm->invert_page_table[lru_idx], lru_idx);
    }
    vm_swap_to_data(vm, vpn, lru_idx);
    page_entry = vm_search_vpn(vm, vpn);
  }
  vm_update_queue(vm, page_entry);
  
  vm->buffer[page_entry * vm->PAGESIZE + offset] = value; //TODO
  // printf("vpn:%d vm0%x val:%x, phy:%d, offset:%d\n", vpn, vm->invert_page_table[0], value, page_entry,offset);
}



__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  for (int i=0; i<input_size;i++){
    int value = vm_read(vm,i);
    results[i+offset] = value;
  }


}

__device__ int vm_search_vpn(VirtualMemory *vm, u32 vpn) {
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if ((vm->invert_page_table[i] & 0x80000000) == 0) {
      if ((vm->invert_page_table[i]>>28) == threadIdx.x) {
        if ((vm->invert_page_table[i] & 0x0fffffff) == vpn) {
          return i;
        }
      }
    }
  }
  return -1 ;
 
}
__device__ void vm_update_pt(VirtualMemory *vm, u32 vpn, int page_entry) {
    vm->invert_page_table[page_entry] = vpn | (threadIdx.x << 28);
}
__device__ void vm_update_queue(VirtualMemory *vm, int page_entry) {
  vm->time_counter++;
  vm->LRU_ARRAY[page_entry] = vm->time_counter;
}

__device__ int vm_get_LRU_idx(VirtualMemory *vm) {
  int min_idx = 0;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (vm->LRU_ARRAY[i] < vm->LRU_ARRAY[min_idx]) {
      min_idx = i;
    }
  }
  return min_idx;
} 


__device__ void vm_swap_to_storage(VirtualMemory *vm, u32 vpn, int page_entry) {
  for (int i = 0; i < vm->PAGESIZE; i++) {
    vm->storage[vpn * vm->PAGESIZE + i] = vm->buffer[page_entry * vm->PAGESIZE + i];
  }
  // vm->invert_page_table[page_entry] = 0x80000000;
}
__device__ void vm_swap_to_data(VirtualMemory *vm, u32 vpn, int page_entry) {
  for (int i = 0; i < vm->PAGESIZE; i++) {
    vm->buffer[page_entry * vm->PAGESIZE + i] = vm->storage[vpn * vm->PAGESIZE + i];
  }
  vm_update_pt(vm, vpn, page_entry);
}