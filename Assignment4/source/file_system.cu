#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  // fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  // super block 4096 byte
  // init free space management
  // 0-127 bytes
  for (int i = 0; i < 1024; i++) {
    fs->SUPERBLOCK[i] = 0;
  }
  // printf("%d\n", fs->SUPERBLOCK[0]);
  // 128-1151 bytes 
  // modification time sort
  // prev: >>10
  // next: &0x3ff 
  // head: 1152 bytes
  // tail: 1153 bytes
  


  // FCB *ptr = (FCB *) &fs->volume[fcb_base];

  for (int i = 0; i < 1024; i++) {
    set_address(&fs->FCBS[i], 0);  
  }

}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
  int empty_block = -1;
  for (int i = 0; i < 1024; i++) {
    if (VALID(fs->FCBS[i].address)) {
        if (cmp_str(fs->FCBS[i].name, s)) {
          if (op == G_READ) {
            SET_READ(i);
          }
          else {
            SET_WRITE(i);
          }
          return i;
        }
    }
    else {
      empty_block = i;
    }
  }

  if (op == G_READ) {
    printf("No such file!!!\n");
    return 0;
  }
  else if (op == G_WRITE) {
    if (empty_block == -1) {
      printf("The file number reaches the limit!!!\n");
      return 0;
    }
    else {
      // fs->SUPERBLOCK[empty_block] = 1;
      // fs->FCBS[empty_block].create_time = gtime++;
      // fs->FCBS[empty_block].modified_time = 0;
      if (gtime == 65535) {
        gtime = sort_by_time(fs->FCBS);
      }
      // empty not anymore
      SET_VALID(fs->FCBS[empty_block].address);
      copy_str(s, fs->FCBS[empty_block].name);
      set_create_time(&fs->FCBS[empty_block], gtime);
      set_modified_time(&fs->FCBS[empty_block], gtime++);
      fs->FCBS[empty_block].size = 0;
      SET_WRITE(empty_block);
      return empty_block;
    }
  }
  else {
    printf("Please input correct op!!!\n");
    return 0;
  }
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
  if (!READ(fp)) {
    printf("No read permission!\n");
    return ; 
  }
  fp = fp & 0x0000ffff;
  if (fs->FCBS[fp].size < size) {
    printf("access size larger than the actul size of the file!!!\n");
    return ;
  }
  read_blocks(fs, get_address(fs->FCBS[fp]), size, output);
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{


  if (!WRITE(fp)) {
    printf("No write permission!\n");
    return ; 
  }
  fp = fp & 0x0000ffff;

  // printf("name%s time: %d\n", fs->FCBS[fp].name, gtime - 1);
  
  int block_need = ceil(size, 32);
  int block_need_old = ceil(fs->FCBS[fp].size, 32);
  if (block_need <= block_need_old) {
    refill_blocks(fs, get_address(fs->FCBS[fp]), fs->FCBS[fp].size, size, input); 
    fs->FCBS[fp].size = size;
  }
  else {
    int address = find_hole(fs->SUPERBLOCK, size);
    if (address != -1) {
      flush_blocks(fs, get_address(fs->FCBS[fp]), block_need_old);
      fill_blocks(fs, address, size, input);
      set_address(&fs->FCBS[fp], address);
      fs->FCBS[fp].size = size;
    }
    else {

      RESET_VALID(fs->FCBS[fp].address);
      compact_blocks(fs);
      SET_VALID(fs->FCBS[fp].address);
      int address = find_hole(fs->SUPERBLOCK, size);
      if (address != -1) {
        flush_blocks(fs, get_address(fs->FCBS[fp]), block_need_old);
        fill_blocks(fs, address, size, input);
        set_address(&fs->FCBS[fp], address);
        fs->FCBS[fp].size = size;
      }
      else {
        printf("no big enough continous space!!!\n");
      }
      return 0;
    }
  }

  if (gtime == 65535) {
    gtime = sort_by_time(fs->FCBS);
    printf("gtime:%d\n", gtime);
  }

  set_modified_time(&fs->FCBS[fp], gtime++);
  return 0;
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
  if (op == LS_D) {
    FCB valid_fcbs[1024];
    int offset = 0;
    for (int i = 0; i < 1024; i++) {
      if (VALID(fs->FCBS[i].address)) {
        valid_fcbs[offset++] = fs->FCBS[i];
      }
    }
    sort_by_date(valid_fcbs, offset);
    print_array_by_date(valid_fcbs, offset);
  }
  else if (op == LS_S) {
    FCB valid_fcbs[1024];
    int offset = 0;
    for (int i = 0; i < 1024; i++) {
      if (VALID(fs->FCBS[i].address)) {
        valid_fcbs[offset++] = fs->FCBS[i];
      }
    }
    sort_by_size(valid_fcbs, offset);
    print_array_by_size(valid_fcbs, offset);


  }
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	if (op == RM) {
    int file = -1;
    for (int i = 0; i < 1024; i++) {
      if (VALID(fs->FCBS[i].address)) {
        if (cmp_str(fs->FCBS[i].name, s)) {
          file = i;
        }
      }
    }

    if (file == - 1) {
      printf("No such file to delete\n");
    }
    else {
      flush_blocks(fs, get_address(fs->FCBS[file]), ceil(fs->FCBS[file].size, 32));
      RESET_VALID(fs->FCBS[file].address);
    }

  }
}