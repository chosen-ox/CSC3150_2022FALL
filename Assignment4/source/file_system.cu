﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  // init free space management
  for (int i = 0; i < 1024; i++) {
    *volume++ = '\0';
  }
  /*
    FCB structrue
      byte 0: create time
      byte 1: last modified
      byte 2: size
      byte 3~22: name 
  */
  int fcb_base = fs->SUPERBLOCK_SIZE;

}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  for (int i = 0; i < size; i++) {
    output[i] = fs->volume[fp+i];
  }
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{

  // for (int i =0; i < 1024; i++) {
  //   printf("%c\n", ptr[i]);
  // }
  for (int i = 0; i < size+1; i++) {
    fs->volume[fp + i] = input[i];
  }
	/* Implement write operation here */
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
  if (op == LS_D) {

  }
  
  else if (op == LS_S) {

  }
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
}
